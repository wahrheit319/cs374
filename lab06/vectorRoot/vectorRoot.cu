/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 *
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 * Extended for use in CS 374 at Calvin University by Joel C. Adams,
 *   adding error-handling, verification, and sequential comparison.
 */

#include <stdio.h>
#include <hip/hip_runtime.h> // CUDA runtime routines (prefixed with "cuda_")
#include <omp.h>

/**
 * CUDA Kernel Device code
 * Computes the vector addition of A and B into C.
 * The 3 vectors have the same number of elements numElements.
 */
__global__ void vectorRoot(const float *A, float *C, const unsigned long numElements)
{
    unsigned long i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = sqrtf(A[i]);
    }
}

/**
 * Utility to clean up error checking
 * It takes a CUDA error code and an error message to print
 *  if the error code is not cudaSuccess
 */
void checkErr(hipError_t err, const char *msg)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "\n*** %s (error code %d: '%s')!\n\n",
                msg, err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * Host main routine
 */
int main(int argc, char **argv)
{
    // CUDA error code to check return values of CUDA calls
    hipError_t err = hipSuccess;

    // Timing variables
    double start, end;
    double time_copy_to_device = 0.0, time_compute = 0.0, time_copy_to_host = 0.0, time_cuda_total = 0.0;
    double time_sequential = 0.0;

    // Get the desired vector length
    unsigned long numElements = 20000; // default value
    if (argc > 1)
    {
        numElements = strtoul(argv[1], 0, 10);
    }

    // Provide initial user feedback...
    printf("\nVector addition of %lu elements using CUDA\n", numElements);

    // Allocate the host input vectors A & B
    size_t size = numElements * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that host allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Unable to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize host input vectors to floats in range 0.0..1.0
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // 1a. Allocate the device input vectors A & B
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    checkErr(err, "Unable to allocate device vector A");
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);
    checkErr(err, "Unable to allocate device vector B");

    // 1.b. Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);
    checkErr(err, "Unable to allocate device vector C");

    // 2. Copy the host input vectors A and B in host memory to the device input vectors in device memory
    start = omp_get_wtime();
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    checkErr(err, "Unable to copy device vector A from host to device");
    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    checkErr(err, "Unable to copy device vector B from host to device");
    end = omp_get_wtime();
    time_copy_to_device = end - start;

    // 3. Launch the CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    start = omp_get_wtime();
    vectorRoot<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, numElements);

    err = hipGetLastError();
    checkErr(err, "Unable to launch vectorAdd kernel");
    end = omp_get_wtime();
    time_compute = end - start;

    // 4. Copy the device result vector in device memory to the host result vector in host memory.
    start = omp_get_wtime();
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    checkErr(err, "Unable to copy vector C from device to host");
    end = omp_get_wtime();
    time_copy_to_host = end - start;

    // Calculate total CUDA time
    time_cuda_total = time_copy_to_device + time_compute + time_copy_to_host;

    // Inside main(), update the verification loop
    const float ACCEPT_THRESHOLD = 1.0e-6;
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(sqrtf(h_A[i]) - h_C[i]) > ACCEPT_THRESHOLD)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    // Free device global memory
    err = hipFree(d_A);
    checkErr(err, "Unable to free device vector A");
    err = hipFree(d_B);
    checkErr(err, "Unable to free device vector B");
    err = hipFree(d_C);
    checkErr(err, "Unable to free device vector C");

    // Repeat the computation sequentially with timing
    float *test_C = (float *)malloc(size);
    if (test_C == NULL)
    {
        fprintf(stderr, "Unable to allocate test-result vector!\n");
        exit(EXIT_FAILURE);
    }

    start = omp_get_wtime();
    // Inside main(), update the sequential computation loop
    for (int i = 0; i < numElements; ++i)
    {
        test_C[i] = sqrtf(h_A[i]);
    }

    end = omp_get_wtime();
    time_sequential = end - start;

    // Verify CUDA against sequential computation
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(test_C[i] - h_C[i]) > ACCEPT_THRESHOLD)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(test_C);

    // Display the timing results
    printf("CUDA Computation Timings:\n");
    printf("\tTime copying to device:\t%lf seconds\n", time_copy_to_device);
    printf("\tTime for computation:\t%lf seconds\n", time_compute);
    printf("\tTime copying to host:\t%lf seconds\n", time_copy_to_host);
    printf("\tTotal CUDA time:\t%lf seconds\n\n", time_cuda_total);

    printf("Sequential Computation Time:\n");
    printf("\tTotal Sequential time:\t%lf seconds\n", time_sequential);

    // Reset the device and exit
    err = hipDeviceReset();
    checkErr(err, "Unable to reset device");

    printf("\nProgram complete\n\n");
    return 0;
}
